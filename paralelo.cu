#include "hip/hip_runtime.h"
/*
    Autor: Bruno de Almeida Zampirom
    Trabalho de implementação de operações sobre matrizes utilizando CUDA

    **Obs: Para printar o valor das matrizes descomentar funções printVET(), printA(), printD(), printVET() nas linhas 89, 98, 107 e 116.
*/
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define TAM 6000
#define escalar 35.5

#define THREADSPERBLOCK 1024
#define BLOCKSPERGRID 188

float A[TAM][TAM], B[TAM][TAM], C[TAM][TAM], D[TAM][TAM], V[TAM], VET[TAM];

float *d_A, *d_B, *d_C, *d_D, *d_V, *d_VET;
size_t size, size_vec;

void printA();
void printC();
void printD();
void printVET();
__global__ void initializationVarCUDA(float *A, float *B, float *C, float *D, float *V, float *VET, int N);
__global__ void multiplyMatrix(float *A, float *B, float *D, int N);
__global__ void multiplyMatrixPerVector(float *B, float *V, float *VET, int N);
__global__ void multiplyMatrixPerEscalar(float *A, int N);
__global__ void someMatrix(float *A, float *B, float *C, int N);

int main()
{
    auto startTotal = high_resolution_clock::now();
    cout << "Starting..." << endl;

    size = TAM * TAM * sizeof(float);
    size_vec = TAM * sizeof(float);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid(BLOCKSPERGRID, BLOCKSPERGRID);

    cout << "Initializing CUDA variables..." << endl;
    auto start = high_resolution_clock::now();

    if (hipMalloc((void**)&d_A, size) != hipSuccess)
        printf("Erro de alocação do vetor d_A\n");
    if (hipMalloc((void**)&d_B, size) != hipSuccess)
        printf("Erro de alocação do vetor d_B\n");
    if (hipMalloc((void**)&d_C, size) != hipSuccess)
        printf("Erro de alocação do vetor d_C\n");
    if (hipMalloc((void**)&d_D, size) != hipSuccess)
        printf("Erro de alocação do vetor d_D\n");
    if (hipMalloc((void**)&d_V, size_vec) != hipSuccess)
        printf("Erro de alocação do vetor d_V\n");
    if (hipMalloc((void**)&d_VET, size_vec) != hipSuccess)
        printf("Erro de alocação do vetor d_VET\n");

    initializationVarCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, d_V, d_VET, TAM);

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time: " << duration.count() / 1000000.0 << " seconds" << endl;

    printf("\nN: %d\nBlocos: %d\nThreads: %d\n", TAM*TAM, (BLOCKSPERGRID * BLOCKSPERGRID), THREADSPERBLOCK);

    cout << endl << "( C = A + B ) - Adding matrix A and B and storing in C..." << endl;
    auto start1 = high_resolution_clock::now();
    someMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, TAM);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    auto stop1 = high_resolution_clock::now();
    auto duration1 = duration_cast<microseconds>(stop1 - start1);
    cout << "Time: " << duration1.count() / 1000000.0  << " seconds" << endl;
    // printC();

    cout << endl << "( D = A * B ) - Multiply matrix A and B and storing in D..." << endl;
    auto start2 = high_resolution_clock::now();
    multiplyMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, TAM);
    hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);
    auto stop2 = high_resolution_clock::now();
    auto duration2 = duration_cast<microseconds>(stop2 - start2);
    cout << "Time: " << duration2.count() / 1000000.0  << " seconds" << endl;
    // printD();

    cout << endl << "( A = A * "<< escalar <<" ) - Multiply matrix A and escalar and storing in A..." << endl;
    auto start3 = high_resolution_clock::now();
    multiplyMatrixPerEscalar<<<blocksPerGrid, threadsPerBlock>>>(d_A, TAM);
    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
    auto stop3 = high_resolution_clock::now();
    auto duration3 = duration_cast<microseconds>(stop3 - start3);
    cout << "Time: " << duration3.count() / 1000000.0 << " seconds" << endl;
    // printA();

    cout << endl << "( VET = V * B ) - Multiply matrix B and vector C and storing in VET..." << endl;
    auto start4 = high_resolution_clock::now();
    multiplyMatrixPerVector<<<THREADSPERBLOCK, BLOCKSPERGRID>>>(d_B, d_V, d_VET, TAM);
    hipMemcpy(VET, d_VET, size_vec, hipMemcpyDeviceToHost);
    auto stop4 = high_resolution_clock::now();
    auto duration4 = duration_cast<microseconds>(stop4 - start4);
    cout << "Time: " << duration4.count() / 1000000.0 << " seconds" << endl;
    // printVET();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_VET);
    hipFree(d_V);

    auto stopTotal = high_resolution_clock::now();
    auto durationTotal = duration_cast<microseconds>(stopTotal - startTotal);
    cout << "\n--------------------------------------------------" << endl;
    cout << "Total algorithm time: " << durationTotal.count() / 1000000.0 << " seconds" << endl;

    return 0;
}

__global__ void multiplyMatrixPerEscalar(float *A, int N) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < N && col < N) {
        A[row * N + col] *= escalar;
    }
}

// Multiplying matrix A and B and storing in C.
__global__ void multiplyMatrix(float *A, float *B, float *D, int N) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;
    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            tmpSum += A[row * N + i] * B[i * N + col];
        }
        D[row * N + col] = tmpSum;
    }
}

// Multiplying matrix A and B and storing in C.
__global__  void multiplyMatrixPerVector(float *B, float *V, float *VET, int N) {
    int tid = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;
    if (tid < N) {
        for (int i = 0; i < N; i++) {
            tmpSum += V[i] * B[i * N + tid];
        }
        VET[tid] = tmpSum;
    }
}

// Adding matrix A and B and storing in C.
__global__ void someMatrix(float *A, float *B, float *C, int N) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < N && col < N) {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

__global__ void initializationVarCUDA(float *A, float *B, float *C, float *D, float *V, float *VET, int N) {
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    if (row < N && col < N) {
        A[row * N + col] = row + col;
        B[row * N + col] = row + col;
        C[row * N + col] = 0;
        D[row * N + col] = 0;
        V[row] = row;
        VET[row] = 0;
    }
}

// Displaying the C matrix.
void printC() {
    bool teste = false;
    for(int i = 0; i < TAM; ++i)
    for(int j = 0; j < TAM; ++j)
    {
        if (i > 0 && j > 0 && C[i][j] == 0) teste = true;
        cout << " " << C[i][j];
        if(j == TAM-1)
            cout << endl;
    }

    if(teste) printf("Ocorreu um erro!");
}

// Displaying the D matrix.
void printD() {
    bool teste = false;
    for(int i = 0; i < TAM; ++i)
    for(int j = 0; j < TAM; ++j)
    {
        if (i > 0 && j > 0 && D[i][j] == 0) teste = true;
        cout << " " << D[i][j];
        if(j == TAM-1)
            cout << endl;
    }
    if(teste) printf("Ocorreu um erro!");
}

// Displaying the A matrix.
void printA() {
    bool teste = false;
    for(int i = 0; i < TAM; ++i)
    for(int j = 0; j < TAM; ++j)
    {
        if (i > 0 && j > 0 && A[i][j] == 0) teste = true;
        cout << " " << A[i][j];
        if(j == TAM-1)
            cout << endl;
    }
    if(teste) printf("Ocorreu um erro!");
}

// Displaying the VET vector.
void printVET() {
    bool teste = false;
    for(int i = 0; i < TAM; ++i) {
        if (i > 0 && VET[i] == 0) teste = true;
        cout << " " << VET[i];
    }
    cout << endl;
    if(teste) printf("Ocorreu um erro!");
}
